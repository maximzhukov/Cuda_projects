#include "hip/hip_runtime.h"
// Made by Maxim Zhukov
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/swap.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/transform_iterator.h>
#include "strided_range.h"
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>

using namespace std;
using namespace thrust;

const unsigned BLOCKS = 1024;
const unsigned THREADS = 1024;

#define __TIME_COUNT__


void throw_on_cuda_error(const hipError_t& code, int itter){
    if(code != hipSuccess){
        string err = hipGetErrorString(code);
        err += ", on iteration: ";
        err += to_string(itter);
        throw runtime_error(err);
    }
}

struct abs_functor : public thrust::unary_function<double, double>{
    __host__ __device__
    double operator()(double elem) const {
        return elem < 0.0 ? -elem : elem;
    }
};

struct abs_comparator{
    abs_functor fabs;

    __host__ __device__ double operator()(double a, double b){
        return fabs(a) < fabs(b);
    }
};

__global__ void gauss_step_L(double* C,  unsigned n, unsigned size, 
                                            unsigned col, double max_elem){
    unsigned thrd_idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x*gridDim.x;

    for(unsigned index = thrd_idx + col + 1; index < n; index += thrd_step){
        C[size*index + col] /= max_elem;
    }
}

__global__ void gauss_step_U(double* C, unsigned n, unsigned size, unsigned col){
    unsigned thrd_idx = threadIdx.x;
    unsigned blck_idx = blockIdx.x;
    unsigned thrd_step = blockDim.x;
    unsigned blck_step = gridDim.x;

    unsigned starting_point_blck = col + 1;
    unsigned starting_point_thrd = (col + 1) - ((col + 1)&255); // start aligned by 256 

    
    for(unsigned i = blck_idx + starting_point_blck; i < n; i += blck_step){
        double coeff = C[i*size + col]; // get coeff

        // first itter may be not full
        unsigned j = thrd_idx + starting_point_thrd;

        if(j > col && j < n){
            C[i*size + j] -= coeff * C[col*size + j];
        }

        for(j += thrd_step; j < n; j+= thrd_step){
            C[i*size + j] -= coeff * C[col*size + j];
        }
    }
}

// 11111111 11111111 11111111 11111111
unsigned get_aligned_size(unsigned n){
    unsigned size = n;
    // 256 = 2^8 =>
    unsigned modulo = n & 255;
    if(modulo){
        size -= modulo;
        size += 256;
    }
    return size;
}

/*
void swap_two_lines(device_vector<double>& matrix, unsigned idx, unsigned jdx, unsigned n, unsigned size){
    thrust::swap_ranges(thrust::device, 
        matrix.begin() + idx * size, 
        matrix.begin() + idx * size + n, 
        matrix.begin() + jdx * size
    );
}
*/

__global__ void swap_lines(double* C, unsigned n, unsigned size, unsigned line1, unsigned line2){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    //unsigned start_point = (col + 1) >> 8;
    for(unsigned index = thrd_idx; index < n; index += thrd_step){
        double temp = C[line1*size + index];
        C[line1*size + index] = C[line2*size + index];
        C[line2*size + index] = temp;
    }
}


int main(){
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(nullptr);

    unsigned n;
    cin >> n;

    unsigned align = get_aligned_size(n);
    // alloc mem to union matrix(see wiki algorithm)
    host_vector<double> h_C(align * n);
    device_vector<double> d_C;
    host_vector<unsigned> h_ansvec(n);

    //host_vector<unsigned> h_p(n);
    //device_vector<unsigned> d_p(n);

    // 256 = 2^8 =>

    // input of matrix
    for(unsigned i = 0; i < n; ++i){
        h_ansvec[i] = i; // init of permutation vector
        for(unsigned j = 0; j < n; ++j){
            cin >> h_C[i*align + j]; 
        }
    }

    // transporting mem to device:
    d_C = h_C;

    // pointer to mem:
    double* raw_C = thrust::raw_pointer_cast(d_C.data());

    #ifdef __TIME_COUNT__
    hipEvent_t start, stop;
    float gpu_time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    #endif

    // compute  LU
    try{
        for(unsigned i = 0; i < n - 1; ++i){
            // create iterator:
            
            strided_range<thrust::device_vector<double>::iterator> range(
                d_C.begin() + i, 
                d_C.end(), 
                align
            ); 
            

            auto it_beg = range.begin();


            auto max_elem = thrust::max_element(
                it_beg + i, it_beg + n, abs_comparator()
            );

            unsigned max_idx = max_elem - it_beg;
            double max_val = *max_elem;

            //cout << "Max elem: " << max_val << endl;

            if(max_idx != i){
                swap_lines<<<BLOCKS, THREADS>>>(raw_C, n, align, i, max_idx);
            }

            gauss_step_L<<<BLOCKS, THREADS>>>(raw_C, n, align, i, max_val);

            throw_on_cuda_error(hipGetLastError(), i);
            throw_on_cuda_error(hipDeviceSynchronize(), i);

            gauss_step_U<<<BLOCKS, THREADS>>>(raw_C, n, align, i);

            h_ansvec[i] = max_idx;

            throw_on_cuda_error(hipGetLastError(), i);
            throw_on_cuda_error(hipDeviceSynchronize(), i);
        }
    }catch(runtime_error& err){
        cout << "ERROR: " << err.what() << endl;
    }

    #ifdef __TIME_COUNT__
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    // open log:
    ofstream log("logs.log", ios::app);
    log << "Fast classic" << endl;
    // threads
    log << BLOCKS << endl;
    // size:
    log << n << endl;
    // time:
    log << gpu_time << endl;
    log.close();
    #endif

    // memcpy from device to host
    h_C = d_C;

    #ifndef __TIME_COUNT__
    // output for matrix:
    cout << std::scientific << std::setprecision(10);
    for(unsigned i = 0; i < n; ++i){
        for(unsigned j = 0; j < n; ++j){
            cout << h_C[i*align + j] << " ";
        }
        cout << endl;
    }
    // output of vector
    for(unsigned i = 0; i < n; ++i){
        cout << h_ansvec[i] << " ";
    }
    cout << endl;
    #endif

    return 0;
}
