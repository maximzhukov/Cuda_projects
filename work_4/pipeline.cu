#include "hip/hip_runtime.h"
// Made by Maxim Zhukov
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <iomanip>
#include <fstream>

using namespace std;
using namespace thrust;

const unsigned BLOCKS = 1024;
const unsigned THREADS = 1024;

#define __TIME_COUNT__

struct abs_functor : public thrust::unary_function<double, double>{
    __host__ __device__
    double operator()(double elem) const {
        return elem < 0.0 ? -elem : elem;
    }
};

struct abs_comparator{
    abs_functor fabs;

    __host__ __device__ double operator()(double a, double b){
        return fabs(a) < fabs(b);
    }
};


void throw_on_cuda_error(const hipError_t& code){
    if(code != hipSuccess){
        throw runtime_error(hipGetErrorString(code));
    }
}

__global__ void gauss_step_L(double* C, unsigned n, unsigned col, double max_elem){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    //unsigned start_pounsigned = (col + 1) >> 8;
    unsigned idx0 = n * col;

    for(unsigned index = thrd_idx + col + 1; index < n; index += thrd_step){
        C[idx0 + index] /= max_elem;
    }
}

__global__ void gauss_step_U(double* C, unsigned n, unsigned col){
    unsigned i_idx = threadIdx.x;
    unsigned j_idx = blockIdx.x;

    unsigned i_step = blockDim.x;
    unsigned j_step = gridDim.x;


    for(unsigned jndex = j_idx + col + 1; jndex < n; jndex += j_step){
        unsigned idx0 = jndex*n;
        double C_jc = C[idx0 + col];
    
        for(unsigned index = i_idx + col + 1; index < n; index += i_step){
            C[idx0 + index] -= C[n*col + index] * C_jc;
        }
    }
}

__global__ void swap_lines(double* C, unsigned n, unsigned line1, unsigned line2){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    //unsigned start_pounsigned = (col + 1) >> 8;
    for(unsigned index = thrd_idx; index < n; index += thrd_step){
        double temp = C[index*n + line1];
        C[index*n + line1] = C[index*n + line2];
        C[index*n + line2] = temp;
    }
}


int main(){
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(nullptr);
    
    unsigned n;
    cin >> n;

    // alloc mem to union matrix(see wiki algorithm)
    //int size = get_allign_size(n);

    double* h_C = (double*) malloc(n * n * sizeof(double));
    unsigned* h_p = (unsigned*) malloc(n * sizeof(unsigned));
    double* d_C;
    throw_on_cuda_error(hipMalloc((void**) &d_C, sizeof(double) * n * n));

    // input of matrix
    for(unsigned i = 0; i < n; ++i){
        h_p[i] = i; // init of permutation vector
        for(unsigned j = 0; j < n; ++j){
            cin >> h_C[j*n + i]; // we store need matrix in  transpose format here for easy thrust search
        }
    }

    // transporting mem to device:
    //d_p = h_ansvec
    throw_on_cuda_error(hipMemcpy(d_C, h_C, sizeof(double) * n * n, hipMemcpyHostToDevice));

    #ifdef __TIME_COUNT__
    hipEvent_t start, stop;
    float gpu_time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    #endif

    // compute  LU
    try{
        for(unsigned i = 0; i < n - 1; ++i){
            // search index of max elem in col
            //auto it_beg = make_transform_iterator(d_C.begin() + i*size, abs_functor());

            auto it_beg = thrust::device_pointer_cast(d_C + i*n);

            auto max_elem = thrust::max_element(it_beg + i, it_beg + n, abs_comparator());

            unsigned max_idx = max_elem - it_beg;
            double max_val = *max_elem;

            if(i != max_idx){
                swap_lines<<<BLOCKS, THREADS>>>(d_C, n, i, max_idx);
                h_p[i] = max_idx;
                //throw_on_cuda_error(hipGetLastError(), i);
                hipDeviceSynchronize();
            }

            gauss_step_L<<<BLOCKS, THREADS>>>(d_C, n, i, max_val);
            //throw_on_cuda_error(hipGetLastError(), i);
            hipDeviceSynchronize();

            gauss_step_U<<<BLOCKS, THREADS>>>(d_C, n, i);
            throw_on_cuda_error(hipGetLastError());
            //throw_on_cuda_error(hipDeviceSynchronize(), i);
        }
    }catch(runtime_error& err){
        cout << "ERROR: " << err.what() << endl;
    }

    #ifdef __TIME_COUNT__
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    // open log:
    ofstream log("logs.log", ios::app);
    log << "Slow transpose" << endl;
    // threads
    log << BLOCKS << endl;
    // size:
    log << n << endl;
    // time:
    log << gpu_time << endl;
    log.close();
    #endif

    throw_on_cuda_error(hipMemcpy(h_C, d_C, sizeof(double) * n * n, hipMemcpyDeviceToHost));
    throw_on_cuda_error(hipFree(d_C));

    #ifndef __TIME_COUNT__
    for(unsigned i = 0; i < n; ++i){
        for(unsigned j = 0; j < n; ++j){
            cout << std::scientific << std::setprecision(10) << h_C[j*n + i] << " ";
        }
        cout << endl;
    }
    // output of vector
    for(unsigned i = 0; i < n; ++i){
        cout << h_p[i] << " ";
    }
    cout << endl;

    #endif

    free(h_C);
    free(h_p);

    return 0;
}
