#include "hip/hip_runtime.h"
// Made by Maxim Zhukov
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>

using namespace std;
using namespace thrust;

const unsigned BLOCKS = 1024;
const unsigned THREADS = 1024;

#define __TIME_COUNT__

struct abs_functor : public thrust::unary_function<double, double>{
    __host__ __device__
    double operator()(double elem) const {
        return elem < 0.0 ? -elem : elem;
    }
};


struct abs_comparator{
    abs_functor fabs;

    __host__ __device__ 
    double operator()(double a, double b){
        return fabs(a) < fabs(b);
    }
};


void throw_on_cuda_error(const hipError_t& code, int itter){
    if(code != hipSuccess){
        string err = hipGetErrorString(code);
        err += ", on iteration: ";
        err += to_string(itter);
        throw runtime_error(err);
    }
}

__global__ void gauss_step_L(double* C, unsigned n, unsigned size, unsigned col, double max_elem){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    //unsigned start_point = (col + 1) >> 8;
    unsigned index = thrd_idx + col + 1 - ((col + 1) & 255);
    unsigned idx0 = size * col;
    
    if(index > col && index < n){
        C[idx0 + index] /= max_elem;
    }

    for(index += thrd_step; index < n; index += thrd_step){
        C[idx0 + index] /= max_elem;
    }
}

__global__ void gauss_step_U(double* C, unsigned n, unsigned size, unsigned col){
    unsigned i_idx = threadIdx.x;
    unsigned j_idx = blockIdx.x;

    unsigned i_step = blockDim.x;
    unsigned j_step = gridDim.x;


    for(unsigned jndex = j_idx + col + 1; jndex < n; jndex += j_step){
        unsigned idx0 = jndex*size;
        double C_jc = C[idx0 + col];

        unsigned index = i_idx + col + 1 - ((col + 1) & 255); 

        if(index > col && index < n){
            //printf("[%d, %d] = %f\n", index, jndex, C[idx0 + index]);
            C[idx0 + index] -= C[size*col + index] * C_jc;
            //printf("[%d, %d] = %f\n", index, jndex, C[idx0 + index]);
        }
    
        for(index += i_step; index < n; index += i_step){
            C[idx0 + index] -= C[size*col + index] * C_jc;
        }
    }
}

__global__ void swap_lines(double* C, unsigned n, unsigned size, unsigned line1, unsigned line2){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    //unsigned start_point = (col + 1) >> 8;
    for(unsigned index = thrd_idx; index < n; index += thrd_step){
        double temp = C[index*size + line1];
        C[index*size + line1] = C[index*size + line2];
        C[index*size + line2] = temp;
    }
}


unsigned get_allign_size(unsigned size){
    unsigned ans = size;
    // 256 = 2^8 =>
    unsigned modulo = size & 255;
    if(modulo){
        ans -= modulo;
        ans += 256;
    }
    return ans;
}

int main(){
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(nullptr);


    unsigned n;
    cin >> n;
    // alloc mem to union matrix(see wiki algorithm)
    unsigned size = get_allign_size(n);

    host_vector<double> h_C(size * n);
    device_vector<double> d_C;
    host_vector<unsigned> h_p(n);

    //host_vector<unsigned> h_p(n);
    //device_vector<unsigned> d_p(n);

    // input of matrix
    for(unsigned i = 0; i < n; ++i){
        h_p[i] = i; // init of permutation vector
        for(unsigned j = 0; j < n; ++j){
            cin >> h_C[j*size + i]; // we store need matrix in  transpose format here for easy thrust search
        }
    }

    // transporting mem to device:
    // memcpy host to device
    d_C = h_C;

    // pointers to mem:
    double* raw_C = thrust::raw_pointer_cast(d_C.data());
    //unsigned* raw_p = thrust::raw_pointer_cast(d_p.data());

    // compute  LU
    #ifdef __TIME_COUNT__
    hipEvent_t start, stop;
    float gpu_time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    #endif

    try{
        for(unsigned i = 0; i < n - 1; ++i){
            // search index of max elem in col
            auto it_beg = d_C.begin() + i*size;


            auto max_elem = thrust::max_element(it_beg + i, it_beg + size, abs_comparator());

            unsigned max_idx = max_elem - it_beg;
            double max_val = *max_elem;

            if(max_idx != i){
                swap_lines<<<BLOCKS, THREADS>>>(raw_C, n, size, i, max_idx);
                h_p[i] = max_idx;
                hipDeviceSynchronize();
            }

            gauss_step_L<<<BLOCKS, THREADS>>>(raw_C, n, size, i, max_val);
            hipDeviceSynchronize();

            gauss_step_U<<<BLOCKS, THREADS>>>(raw_C, n, size, i);
            throw_on_cuda_error(hipGetLastError(), i);
        }
    }catch(runtime_error& err){
        cout << "ERROR: " << err.what() << endl;
    }

    #ifdef __TIME_COUNT__
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    // open log:
    ofstream log("logs.log", ios::app);
    log << "Fast transpose" << endl;
    // threads
    log << BLOCKS << endl;
    // size:
    log << n << endl;
    // time:
    log << gpu_time << endl;
    log.close();
    #endif

    

    // memcpy device to host
    h_C = d_C;

    #ifndef __TIME_COUNT__
    // output for matrix:
    cout << std::scientific << std::setprecision(10);
    for(unsigned i = 0; i < n; ++i){
        for(unsigned j = 0; j < n; ++j){
            cout << h_C[j*size + i] << " ";
        }
        cout << endl;
    }
    // output of vector
    for(unsigned i = 0; i < n; ++i){
        cout << h_p[i] << " ";
    }
    cout << endl;
    #endif

    return 0;
}
